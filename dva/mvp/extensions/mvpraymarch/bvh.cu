#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
// All rights reserved.
// 
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#include <cmath>
#include <cstdio>
#include <functional>
#include <map>

#include "hip/hip_vector_types.h"

#include "cudadispatch.h"

#include "primtransf.h"

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__ unsigned int expand_bits(unsigned int v) {
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__device__ unsigned int morton3D(float x, float y, float z) {
    x = fminf(fmaxf(x * 1024.0f, 0.0f), 1023.0f);
    y = fminf(fmaxf(y * 1024.0f, 0.0f), 1023.0f);
    z = fminf(fmaxf(z * 1024.0f, 0.0f), 1023.0f);
    unsigned int xx = expand_bits((unsigned int)x);
    unsigned int yy = expand_bits((unsigned int)y);
    unsigned int zz = expand_bits((unsigned int)z);
    return xx * 4 + yy * 2 + zz;
}

template<typename PrimTransfT>
__global__ void compute_morton_kernel(
        int N, int K,
        typename PrimTransfT::Data data,
        int * code
        ) {
    const int count = N * K;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
        const int k = index % K;
        const int n = index / K;

        //float4 c = center[n * K + k];
        float3 c = data.get_center(n, k);
        code[n * K + k] = morton3D(c.x, c.y, c.z);
    }
}

__forceinline__ __device__ int delta(int* sortedcodes, int x, int y, int K) {
	if (x >= 0 && x <= K - 1 && y >= 0 && y <= K - 1) {
        return sortedcodes[x] == sortedcodes[y] ?
            32 + __clz(x ^ y) :
            __clz(sortedcodes[x] ^ sortedcodes[y]);
    }
	return -1;
}

__forceinline__ __device__ int sign(int x) {
	return (int)(x > 0) - (int)(x < 0);
}

__device__ int find_split(
       int* sortedcodes,
       int first,
       int last,
       int K) {
    float commonPrefix = delta(sortedcodes, first, last, K);
    int split = first;
    int step = last - first;

    do {
        step = (step + 1) >> 1; // exponential decrease
        int newSplit = split + step; // proposed new position

        if (newSplit < last) {
            int splitPrefix = delta(sortedcodes, first, newSplit, K);
            if (splitPrefix > commonPrefix) {
                split = newSplit; // accept proposal
            }
        }
    } while (step > 1);

    return split;
}

__device__ int2 determine_range(int* sortedcodes, int K, int idx) {
    int d = sign(delta(sortedcodes, idx, idx + 1, K) - delta(sortedcodes, idx, idx - 1, K));
    int dmin = delta(sortedcodes, idx, idx - d, K);
    int lmax = 2;
    while (delta(sortedcodes, idx, idx + lmax * d, K) > dmin) {
        lmax = lmax * 2;
    }

    int l = 0;
    for (int t = lmax / 2; t >= 1; t /= 2) {
        if (delta(sortedcodes, idx, idx + (l + t)*d, K) > dmin) {
            l += t;
        }
    }

    int j = idx + l*d;
    int2 range;
    range.x = min(idx, j);
    range.y = max(idx, j);

    return range;
}

__global__ void build_tree_kernel(
        int N, int K,
        int * sortedcodes,
        int2 * nodechildren,
        int * nodeparent) {
    const int count = N * (K + K - 1);
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
        const int k = index % (K + K - 1);
        const int n = index / (K + K - 1);

        if (k >= K - 1) {
            // leaf
            nodechildren[n * (K + K - 1) + k] = make_int2(-(k - (K - 1)) - 1, -(k - (K - 1)) - 2);
        } else {
            // internal node

            // find out which range of objects the node corresponds to
            int2 range = determine_range(sortedcodes + n * K, K, k);
            int first = range.x;
            int last = range.y;

            // determine where to split the range
            int split = find_split(sortedcodes + n * K, first, last, K);

            // select childA
            int childa = split == first ? (K - 1) + split : split;

            // select childB
            int childb = split + 1 == last ? (K - 1) + split + 1 : split + 1;

            // record parent-child relationships
            nodechildren[n * (K + K - 1) + k] = make_int2(childa, childb);
            nodeparent[n * (K + K - 1) + childa] = k;
            nodeparent[n * (K + K - 1) + childb] = k;
        }
    }
}

template<typename PrimTransfT>
__global__ void compute_aabb_kernel(
        int N, int K,
        typename PrimTransfT::Data data,
        int * sortedobjid,
        int2 * nodechildren,
        int * nodeparent,
        float3 * nodeaabb,
        int * atom) {
    const int count = N * K;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
        const int k = index % K;
        const int n = index / K;

        // compute BBOX for leaf
        int kk = sortedobjid[n * K + k];

        float3 pmin;
        float3 pmax;
        data.compute_aabb(n, kk, pmin, pmax);

        nodeaabb[n * (K + K - 1) * 2 + ((K - 1) + k) * 2 + 0] = pmin;
        nodeaabb[n * (K + K - 1) * 2 + ((K - 1) + k) * 2 + 1] = pmax;

        int node = nodeparent[n * (K + K - 1) + ((K - 1) + k)];

        while (node != -1 && atomicCAS(&atom[n * (K - 1) + node], 0, 1) == 1) {
            int2 children = nodechildren[n * (K + K - 1) + node];
            float3 laabbmin = nodeaabb[n * (K + K - 1) * 2 + children.x * 2 + 0];
            float3 laabbmax = nodeaabb[n * (K + K - 1) * 2 + children.x * 2 + 1];
            float3 raabbmin = nodeaabb[n * (K + K - 1) * 2 + children.y * 2 + 0];
            float3 raabbmax = nodeaabb[n * (K + K - 1) * 2 + children.y * 2 + 1];

            float3 aabbmin = fminf(laabbmin, raabbmin);
            float3 aabbmax = fmaxf(laabbmax, raabbmax);

            nodeaabb[n * (K + K - 1) * 2 + node * 2 + 0] = aabbmin;
            nodeaabb[n * (K + K - 1) * 2 + node * 2 + 1] = aabbmax;

            node = nodeparent[n * (K + K - 1) + node];

            __threadfence();
        }
    }
}

void compute_morton_cuda(
        int N, int K,
        float * primpos,
        int * code,
        int algorithm,
        hipStream_t stream) {
    int count = N * K;
    int blocksize = 512;
    int gridsize = (count + blocksize - 1) / blocksize;

    std::shared_ptr<PrimTransfDataBase> primtransf_data;
    primtransf_data = std::make_shared<PrimTransfSRT::Data>(PrimTransfSRT::Data{
            PrimTransfDataBase{},
            K, (float3*)primpos, nullptr,
            K * 3, nullptr, nullptr,
            K, nullptr, nullptr});

    std::map<int, std::function<void(dim3, dim3, hipStream_t, int, int, std::shared_ptr<PrimTransfDataBase>, int*)>> dispatcher = {
      { 0, make_cudacall(compute_morton_kernel<PrimTransfSRT>) }
    };

    auto iter = dispatcher.find(min(0, algorithm));
    if (iter != dispatcher.end()) {
        (iter->second)(
            dim3(gridsize), dim3(blocksize), stream,
            N, K,
            primtransf_data,
            code);
    }
}

void build_tree_cuda(
        int N, int K,
        int * sortedcode,
        int * nodechildren,
        int * nodeparent,
        hipStream_t stream) {
    int count = N * (K + K - 1);
    int nthreads = 512;
    int nblocks = (count + nthreads - 1) / nthreads;
    build_tree_kernel<<<nblocks, nthreads, 0, stream>>>(
            N, K,
            sortedcode,
            reinterpret_cast<int2 *>(nodechildren),
            nodeparent);
}

void compute_aabb_cuda(
        int N, int K,
        float * primpos,
        float * primrot,
        float * primscale,
        int * sortedobjid,
        int * nodechildren,
        int * nodeparent,
        float * nodeaabb,
        int algorithm,
        hipStream_t stream) {
    int * atom;
    hipMalloc(&atom, N * (K - 1) * 4);
    hipMemset(atom, 0, N * (K - 1) * 4);

    int count = N * K;
    int blocksize = 512;
    int gridsize = (count + blocksize - 1) / blocksize;

    std::shared_ptr<PrimTransfDataBase> primtransf_data;
    primtransf_data = std::make_shared<PrimTransfSRT::Data>(PrimTransfSRT::Data{
            PrimTransfDataBase{},
            K, (float3*)primpos, nullptr,
            K * 3, (float3*)primrot, nullptr,
            K, (float3*)primscale, nullptr});

    std::map<int, std::function<void(dim3, dim3, hipStream_t, int, int, std::shared_ptr<PrimTransfDataBase>, int*, int2*, int*, float3*, int*)>> dispatcher = {
      { 0, make_cudacall(compute_aabb_kernel<PrimTransfSRT>) }
    };
    
    auto iter = dispatcher.find(min(0, algorithm));
    if (iter != dispatcher.end()) {
        (iter->second)(
            dim3(gridsize), dim3(blocksize), stream,
            N, K,
            primtransf_data,
            sortedobjid,
            reinterpret_cast<int2 *>(nodechildren),
            nodeparent,
            reinterpret_cast<float3 *>(nodeaabb),
            atom);
    }

    hipFree(atom);
}
